#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "FreeImage.h"

#define BPP 24 // Since we're outputting three 8 bit RGB values

using namespace std;

__global__ void gris_niveau(int * d_img, int *img, unsigned width, unsigned height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  int idx = ((y * width) + x) * 3;
  
  // compute weighted avrage, to get gray scale value of pixel
  int grey = d_img[idx+0]*0.299 + d_img[idx+1]*0.587 + d_img[idx+2]*0.114;

  if ((x< width) && (y < height))
  {
         // the three colors have the same value of gray
   	 d_img[idx + 0] = grey;
      	 d_img[idx + 1] = grey;
      	 d_img[idx + 2] = grey;
  }

}

int main(int argc, char** argv)
{

  FreeImage_Initialise();
  const char *PathName = "img.jpg";
  const char *PathDest = "new_img.png";
  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);

  FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if(! bitmap )
    exit( 1 ); //WTF?! We can't even allocate images ? Die !

  unsigned width  = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch  = FreeImage_GetPitch(bitmap);

  fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

  int size = sizeof(unsigned int) * 3 * width * height;

  unsigned int *img = (unsigned int*) malloc(size);

  // convert image to array
  BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  int *d_img,*d_img1;
  hipMalloc((void**)&d_img,size);
  hipMemcpy(d_img, img,size,hipMemcpyHostToDevice);
  hipMalloc((void**)&d_img1,size);
  hipMemcpy(d_img1, img,size,hipMemcpyHostToDevice);

  dim3 dimGrid (128,72,1);
  dim3 dimBlock (30,30,1);
  
  // call kernel to convert image to gray scale
  gris_niveau<<<dimGrid, dimBlock>>>(d_img,d_img1, width, height);

  hipMemcpy(img,d_img,size,hipMemcpyDeviceToHost);

  // save image
  bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
    printf("Image successfully saved !\n");
  FreeImage_DeInitialise(); //Cleanup !

  hipFree(d_img);
  hipFree(d_img1);

  free(img);

  return 0;
}
